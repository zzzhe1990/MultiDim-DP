#include "hip/hip_runtime.h"
#include <sys/time.h>
#include "type.h"
#include <iostream>

using namespace std;

__global__ void mainIteration(int m, UINT64 total_weight, int blockSize, int gridSize, 
			      int idxOffset, int profit, int *dev_array1, int *dev_array2,
			      int *dev_cap, int *dev_weight, int item){
	int thread = blockIdx.x * blockDim.x + threadIdx.x;
	//for all possible MKP vector capacities between 0 and cap
	
	for (UINT64 idx = thread; idx < total_weight; idx += (blockSize*gridSize) ){
		
		int maxprofit = dev_array1[idx];
		int td = idx;
		int di, test = 1;
		
		for (int i=0; i<m; i++){
			di = idx % dev_cap[i];
			if (di < dev_weight[i]){
				test = 0;
				break;
			}	
			td = td / dev_cap[i];
		}		

		if(test){
			//T_(k)(d) = max(T_(k-1)(d), T_(k-1)(d-w_k)+p_k);
			//d-w_k
			maxprofit = max(maxprofit, dev_array1[idx - idxOffset] + profit);
		}
		dev_array2[idx] = maxprofit;
	}
}

__global__ void mainIteration1(int m, UINT64 total_weight, int blockSize, int gridSize, 
			      int idxOffset, int profit, int *dev_array1, int *dev_array2,
			      int *dev_cap, int *dev_weight, int item){
	int thread = blockIdx.x * blockDim.x + threadIdx.x;
	//for all possible MKP vector capacities between 0 and cap
	
	for (UINT64 idx = thread; idx < total_weight; idx += (blockSize*gridSize) ){
		
		int maxprofit = dev_array1[idx];
		
		if (idx >= dev_weight[0]){
			//T_(k)(d) = max(T_(k-1)(d), T_(k-1)(d-w_k)+p_k);
			//d-w_k
			maxprofit = max(maxprofit, dev_array1[idx - idxOffset] + profit);
		}
		dev_array2[idx] = maxprofit;
	}
}

__global__ void mainIteration2(int m, UINT64 total_weight, int blockSize, int gridSize, 
			      int idxOffset, int profit, int *dev_array1, int *dev_array2,
			      int *dev_cap, int *dev_weight, int item){
	int thread = blockIdx.x * blockDim.x + threadIdx.x;
	//for all possible MKP vector capacities between 0 and cap
	
	for (UINT64 idx = thread; idx < total_weight; idx += (blockSize*gridSize) ){
		
		int maxprofit = dev_array1[idx];
		int d0, d1;
		
		d0 = idx % dev_cap[0];
		d1 = idx / dev_cap[0];

		if(d0>=dev_weight[0] && d1>= dev_weight[1]){
			maxprofit = max(maxprofit, dev_array1[idx - idxOffset] + profit);
		}
		dev_array2[idx] = maxprofit;
	}
}

__global__ void mainIteration3(int m, UINT64 total_weight, int blockSize, int gridSize, 
			      int idxOffset, int profit, int *dev_array1, int *dev_array2,
			      int *dev_cap, int *dev_weight, int item){
	int thread = blockIdx.x * blockDim.x + threadIdx.x;
	//for all possible MKP vector capacities between 0 and cap
	
	for (UINT64 idx = thread; idx < total_weight; idx += (blockSize*gridSize) ){
		
		int maxprofit = dev_array1[idx];
		int d0, d1, d2, d;
		d0 = idx % dev_cap[0];
		d = idx / dev_cap[0];
		d1 = d % dev_cap[1];
		d2 = d / dev_cap[1];
			
		if(d0>=dev_weight[0] && d1>=dev_weight[1] && d2>=dev_weight[2]){
			maxprofit = max(maxprofit, dev_array1[idx - idxOffset] + profit);
		}
		dev_array2[idx] = maxprofit;
	}
}

__global__ void mainIteration4(int m, UINT64 total_weight, int blockSize, int gridSize, 
			      int idxOffset, int profit, int *dev_array1, int *dev_array2,
			      int *dev_cap, int *dev_weight, int item){
	int thread = blockIdx.x * blockDim.x + threadIdx.x;
	//for all possible MKP vector capacities between 0 and cap
	
	for (UINT64 idx = thread; idx < total_weight; idx += (blockSize*gridSize) ){
		
		int maxprofit = dev_array1[idx];
		int d0, d1, d2, d3;
		int d = idx / dev_cap[0];
		d0 = idx % dev_cap[0];
		d1 = d % dev_cap[1];
		d = d / dev_cap[1];
		d2 = d % dev_cap[2];
		d3 = d / dev_cap[2];

		if(d0>=dev_weight[0] && d1>=dev_weight[1] && d2>=dev_weight[2] && d3>=dev_weight[3]){
			maxprofit = max(maxprofit, dev_array1[idx - idxOffset] + profit);
		}
		dev_array2[idx] = maxprofit;
	}
}
/*
__global__ void mainIteration(int m, UINT64 total_weight, int blockSize, int gridSize, 
			      int idxOffset, int profit, int *dev_array1, int *dev_array2,
			      int *dev_cap, int *dev_weight, int item){
	int thread = blockIdx.x * blockDim.x + threadIdx.x;
	//for all possible MKP vector capacities between 0 and cap
	
	for (UINT64 idx = thread; idx < total_weight; idx += (blockSize*gridSize) ){
		
		int maxprofit = dev_array1[idx];
		int td = dev_array1[idx];
		int di, test = 1;
		
		for (int i=0; i<m; i++){
			di = idx % dev_cap[i];
			if (di < dev_weight[item * m + i]){
				test = 0;
				break;
			}	
			td = td / dev_cap[i];
		}		

		if(test){
			//T_(k)(d) = max(T_(k-1)(d), T_(k-1)(d-w_k)+p_k);
			//d-w_k
				maxprofit = max(maxprofit, dev_array1[idx - idxOffset] + profit);
		}
	}
}
*/

int MKPoffset(int *weight, int *cap, int m){
	int offset = 0;
	int ww = 1;
	for (int i=0; i<m; i++){
		offset += (weight[i] * ww);
		ww *= cap[i];
	}

	return offset;
}

int DPiteration(int m, int n, int *weight, int *profit, int *cap){
	
	struct timeval tbegin, tend;	
	int maxvalue;
	//MKP is a table consist of all constraints and items. Constraints includes no-constraint; item includes 0 item.
	UINT64 total_weight = 1;
	for (int i=0; i<m; i++){
		total_weight *= (UINT64)(cap[i]+1);
	}
	
	int blockSize, gridSize;
	blockSize = 512;
	gridSize = 16;

	int *dev_array1 = 0, *dev_array2 = 0;
	int *dev_cap = 0, *dev_weight = 0;
	
	hipMalloc((void**)dev_array1, total_weight*sizeof(int) );
	hipMalloc((void**)dev_array2, total_weight*sizeof(int) );
	hipMalloc((void**)dev_cap, m*sizeof(int) );
	hipMalloc((void**)dev_weight, n*m*sizeof(int) );
	hipMemset(dev_array1, 0, total_weight*sizeof(int));
	hipMemset(dev_array2, 0, total_weight*sizeof(int));	
	hipMemcpy(dev_cap, cap, m * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_weight, weight, n * m * sizeof(int), hipMemcpyHostToDevice);

	gettimeofday(&tbegin, NULL);
	
	for (int k=0; k<n; k++){
		int idxOffset = MKPoffset(&weight[k*m], cap, m);

		switch(m){	
			case 1: mainIteration1<<<gridSize, blockSize>>>(m, total_weight, blockSize, gridSize, 
						       idxOffset, profit[k], dev_array1, dev_array2, 
						       dev_cap, &dev_weight[k], k);
			break;
			case 2: mainIteration2<<<gridSize, blockSize>>>(m, total_weight, blockSize, gridSize, 
						       idxOffset, profit[k], dev_array1, dev_array2, 
						       dev_cap, &dev_weight[k+k], k);
			break;
			case 3: mainIteration3<<<gridSize, blockSize>>>(m, total_weight, blockSize, gridSize, 
						       idxOffset, profit[k], dev_array1, dev_array2, 
						       dev_cap, &dev_weight[k+k+k], k);
			break;
			case 4: mainIteration4<<<gridSize, blockSize>>>(m, total_weight, blockSize, gridSize, 
						       idxOffset, profit[k], dev_array1, dev_array2, 
						       dev_cap, &dev_weight[k*4], k);
			break;
			default: mainIteration<<<gridSize, blockSize>>>(m, total_weight, blockSize, gridSize, 
						       idxOffset, profit[k], dev_array1, dev_array2, 
						       dev_cap, &dev_weight[k*m], k);
			break;
		}

		hipDeviceSynchronize();
		int *temp = dev_array1;
		dev_array1 = dev_array2;
		dev_array2 = temp;
	}
	
	gettimeofday(&tend, NULL);

	cout << "DP iteration on GPU run time: " <<(tend.tv_usec - tbegin.tv_usec)/1E6 << " second." << endl;
		
	hipMemcpy(&maxvalue, &dev_array2[total_weight-1], sizeof(int), hipMemcpyDeviceToHost);
	
	hipFree(dev_array1);
	hipFree(dev_array2);
	hipFree(dev_cap);
	hipFree(dev_weight);
	
	return maxvalue;
}


