#include "hip/hip_runtime.h"
#include <sys/time.h>
#include "type.h"
#include <iostream>

using namespace std;

__global__ void mainIteration(int m, UINT64 total_weight, int blockSize, int gridSize, 
			      int idxOffset, int profit, int *dev_array1, int *dev_array2,
			      int *dev_cap, int *dev_weight, int item){
	int thread = blockIdx.x * blockDim.x + threadIdx.x;
	//for all possible MKP vector capacities between 0 and cap
	
	for (UINT64 idx = thread; idx < total_weight; idx += (blockSize*gridSize) ){
		
		int maxprofit = dev_array1[idx];
		int td = dev_array1[idx];
		int di, test = 1;
		
		for (int i=0; i<m; i++){
			di = idx % dev_cap[i];
			if (di < dev_weight[item * m + i]){
				test = 0;
				break;
			}	
			td = td / dev_cap[i];
		}		

		if(test){
			//T_(k)(d) = max(T_(k-1)(d), T_(k-1)(d-w_k)+p_k);
			//d-w_k
				maxprofit = max(maxprofit, dev_array1[idx - idxOffset] + profit);
		}
	}
}

int MKPoffset(int *weight, int *cap, int m){
	int offset = 0;
	int ww = 1;
	for (int i=0; i<m; i++){
		offset += (weight[i] * ww);
		ww *= cap[i];
	}

	return offset;
}

int DPiteration(int m, int n, int *weight, int *profit, int *cap){
	
	struct timeval tbegin, tend;	
	int maxvalue;
	//MKP is a table consist of all constraints and items. Constraints includes no-constraint; item includes 0 item.
	UINT64 total_weight = 1;
	for (int i=0; i<m; i++){
		total_weight *= (UINT64)(cap[i]+1);
	}
	
	int blockSize, gridSize;
	blockSize = 512;
	gridSize = 16;

	int *dev_array1 = 0, *dev_array2 = 0;
	int *dev_cap = 0, *dev_weight = 0;
	
	hipMalloc((void**)dev_array1, total_weight*sizeof(int) );
	hipMalloc((void**)dev_array2, total_weight*sizeof(int) );
	hipMalloc((void**)dev_cap, m*sizeof(int) );
	hipMalloc((void**)dev_weight, n*m*sizeof(int) );
	hipMemset(dev_array1, 0, total_weight*sizeof(int));
	hipMemset(dev_array2, 0, total_weight*sizeof(int));	
	hipMemcpy(dev_cap, cap, m * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_weight, weight, n * m * sizeof(int), hipMemcpyHostToDevice);

	gettimeofday(&tbegin, NULL);
	
	for (int k=0; k<n; k++){
		int idxOffset = MKPoffset(&weight[k*m], cap, m);

		mainIteration<<<gridSize, blockSize>>>(m, total_weight, blockSize, gridSize, 
						       idxOffset, profit[k], dev_array1, dev_array2, 
						       dev_cap, dev_weight, k);
		hipDeviceSynchronize();
		int *temp = dev_array1;
		dev_array1 = dev_array2;
		dev_array2 = temp;
	}
	
	gettimeofday(&tend, NULL);

	cout << "DP iteration on GPU run time: " <<(tend.tv_usec - tbegin.tv_usec)/1E6 << " second." << endl;
		
	hipMemcpy(&maxvalue, &dev_array2[total_weight-1], sizeof(int), hipMemcpyDeviceToHost);
	
	hipFree(dev_array1);
	hipFree(dev_array2);
	hipFree(dev_cap);
	hipFree(dev_weight);
	
	return maxvalue;
}


